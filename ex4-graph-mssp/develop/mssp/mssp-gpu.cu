#include "hip/hip_runtime.h"
/*
  to-do : 1. change the matrix order to column major and
             -> change matrix 
	     -> change indexing
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <random>

#include "csr.hpp"

//__________________________________________________________________
// device function for the indexing
// rather a lambda?
__device__ unsigned int crd2idx(unsigned int batch,
				unsigned int batchsize,
				unsigned int v) {
  return batch * batchsize + v;
}

//__________________________________________________________________
// produce a 2d grid based on the sizes
dim3 get_grid(unsigned int x, unsigned int y, dim3 block_2d) {
  dim3 grid_2d((x + (block_2d.x - 1)) / block_2d.x,
	       (y + (block_2d.y - 1)) / block_2d.y);
  return grid_2d;
}

//__________________________________________________________________
// produce a grid_block based on the sizes
/*
dim3 get_block(unsigned int x, unsigned int y) {  
}
*/
//__________________________________________________________________
// gpu function to run the multiple source bellman ford
// 1. use array of things indices to be run
// 2. loop over indices to be run
// 3. give back array of the ones that changed
/*
__global__ void bf_iteration(int           n,
			     unsigned int  batchsize,
			     unsigned int *csr_index,
			     unsigned int *csr_cols,
			     float        *csr_weights,
			     float        *d,
			     float        *d_new,
			     unsigned int *ind,
			     int          *result) {
  auto thisThread = blockIdx.x * blockDim.x + threadIdx.x;
  auto numThreads = gridDim.x + blockDim.x;

  // loop over all the batches that need to be done
  for (unsigned int batch = 0; batch < batchsize; ++batch) {
    bool changes = false;
    auto idx = ind[batch];
    for (unsigned int v = thisThread; v < n; v += numThreads) {
      float dist = d[crd2idx(idx, batchsize, v)];
      for(unsigned int i = csr_index[v]; i < csr_index[v + 1]; ++i) {
	auto u = csr_cols[i];
	auto weight = csr_weights[i];
	
	if(dist > d[crd2idx(idx, batchsize, u)] + weight) {
	  dist = d[crd2idx(idx, batchsize, u)] + weight;
	  changes = true;
	}
      }
      d_new[crd2idx(idx, batchsize, v)] = dist;
    }
    // check if a certain batch changed
    if (changes) {
      result[idx] = 1;
    }
  }
}
*/

__global__ void bf_iteration_2d(int           n,
				unsigned int  batchsize,
				unsigned int *csr_index,
				unsigned int *csr_cols,
				float        *csr_weights,
				float        *d,
				float        *d_new,
				int          *result) {
  auto thread_x = blockIdx.x * blockDim.x + threadIdx.x;
  auto thread_y = blockIdx.y * blockDim.y + threadIdx.y;
  auto n_threads_x = gridDim.x + blockDim.x;
  auto n_threads_y = gridDim.y + blockDim.y;
  
  // loop over all the batches that need to be done
  bool changes = false;
  for (unsigned int batch = thread_y; batch < batchsize; batch += n_threads_y) {
    for (unsigned int v = thread_x; v < n; v += n_threads_x) {
      float dist = d[crd2idx(batch, batchsize, v)];
      for(unsigned int i = csr_index[v]; i < csr_index[v + 1]; ++i) {
	auto u = csr_cols[i];
	auto weight = csr_weights[i];
	
	if(dist > d[crd2idx(batch, batchsize, u)] + weight) {
	  dist = d[crd2idx(batch, batchsize, u)] + weight;
	  changes = true;
	}
      }
      d_new[crd2idx(batch, batchsize, v)] = dist;
    }
    // check if a certain batch changed
  }
  if (changes) {
    *result = 1;
  }
}

//___________________________________________________________________
// run the bf stuff
void run_bf(const csr_matrix                &tr,
	    unsigned int                     batchsize,
	    const std::vector<unsigned int> &sources) {
  // 1.0. allocate memory matrix and move to gpu
  unsigned int *csr_index;
  unsigned int *csr_cols;
  float        *csr_weights;

  hipMalloc(&csr_index, (tr.n + 1) * sizeof(unsigned int));
  hipMalloc(&csr_cols,      tr.nnz * sizeof(unsigned int));
  hipMalloc(&csr_weights,   tr.nnz * sizeof(float));
  
  hipMemcpy(csr_index,   tr.ind.data(), (tr.n + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(csr_cols,    tr.cols.data(),    tr.nnz * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(csr_weights, tr.weights.data(), tr.nnz * sizeof(unsigned int), hipMemcpyHostToDevice);

  // 1.1 allocate memory distances and move to gpu
  float *d;
  float *d_new;
  int   *result;
  hipMalloc(&d,      batchsize * tr.n * sizeof(float));
  hipMalloc(&d_new,  batchsize * tr.n * sizeof(float));
  hipMalloc(&result, /*batchsize * */   sizeof(int));

  std::vector <float> initial;
  initial.resize(tr.n * batchsize);  
  std::fill(initial.begin(), initial.end(), FLT_MAX);
  for (std::size_t b = 0; b < batchsize; ++b) {
    initial[b*batchsize + sources[b]] = 0;
  }
  
  hipMemcpy(d, initial.data(), tr.n * batchsize * sizeof(float), hipMemcpyHostToDevice);

  /*
  // 2. loop over all the problems until they are all solved
  // controll array c for the indices that did change
  // array of indices to run over
  unsigned int *c, *ind_host, *ind_dev;
  c = (unsigned int*) malloc (batchsize * sizeof(unsigned int));
  ind_host = (unsigned int*) malloc (batchsize * sizeof(unsigned int));
  for (unsigned int i = 0; i < batchsize; ++i) {
    ind_host[i] = i;
  }
  hipMalloc(&ind_dev, batchsize*sizeof(unsigned int));
  
  unsigned int num_blocks = (tr.n + 255) / 256;
  unsigned int to_solve = batchsize;
  while(true) {
    hipMemset(result,  0, batchsize*sizeof(int));
    hipMemcpy(ind_dev, ind_host, batchsize*sizeof(int), hipMemcpyHostToDevice);
    bf_iteration<<<num_blocks, 256>>>(tr.n, to_solve,
				      csr_index, csr_cols, csr_weights,
				      d, d_new, ind_dev, result);
    
    // check for iteration and decide which ones should be iterated again
    hipMemcpy(c, result, batchsize*sizeof(int), hipMemcpyDeviceToHost);
    std::size_t cnt = 0;
    for (std::size_t i = 0; i < batchsize; ++i) {
      if (!c[i]) {
	ind_host[cnt] = i;
	++cnt;
      }
    }
    to_solve = cnt;
    if (cnt == batchsize)
      break;
    std::swap(d, d_new);
  }
  */
  // 2d strategy
  dim3 block_2d(1024, 1024);
  dim3 grid_2d = get_grid(tr.n, batchsize, block_2d);
  
  while(true) {
    hipMemset(result, 0, sizeof(int));
    bf_iteration_2d<<<grid_2d, block_2d>>>(tr.n, batchsize,
					   csr_index, csr_cols, csr_weights,
					   d, d_new, result);
    unsigned int c;
    hipMemcpy(&c, result, sizeof(int), hipMemcpyDeviceToHost);
    if (!c)
      break;
    std::swap(d, d_new);
  }
  
  // 4. free memory
  hipFree(csr_index);
  hipFree(csr_cols);
  hipFree(csr_weights);
  hipFree(d);
  hipFree(d_new);
  hipFree(result);
  // hipFree(ind_dev);
  
  // free(c);
  // free(ind_host);
}

//___________________________________________________________________
// int main(int argc, char** argv)
int main(int argc, char **argv) {
  if(argc != 3)
    throw std::runtime_error("Expected instance and batch size as argument");
  
  unsigned int batchsize = std::atoi(argv[2]);
  
  std::mt19937 prng{42};
  std::uniform_real_distribution<float> weight_distrib{0.0f, 1.0f};
  
  // Load the graph.
  std::cout << "algo: " << "bf_gpu" << std::endl;
  std::string instance(argv[1]);
  std::size_t npos = instance.find_last_of("/");
  instance = instance.substr(npos+1);
  std::cout << "instance: " << instance << std::endl;
  std::cout << "batchsize: " << batchsize << std::endl;
  
  std::ifstream ins(argv[1]);
  std::vector<std::tuple<unsigned int, unsigned int, float>> cv;
  
  auto io_start = std::chrono::high_resolution_clock::now();
  read_graph_unweighted(ins, [&] (unsigned int u, unsigned int v) {
      // Generate a random edge weight in [a, b).
      cv.push_back({u, v, weight_distrib(prng)});
    });
  
  auto mat = coordinates_to_csr(std::move(cv));
  auto t_io = std::chrono::high_resolution_clock::now() - io_start;
  
  std::cout << "time_io: "
	    << std::chrono::duration_cast<std::chrono::milliseconds>(t_io).count() << std::endl;
  std::cout << "n_nodes: " << mat.n << std::endl;
  std::cout << "n_edges: " << mat.nnz << std::endl;
  
  auto tr = transpose(std::move(mat));
  
  // Generate random sources.
  std::uniform_int_distribution<unsigned int> s_distrib{0, mat.n - 1};
  std::vector<unsigned int> sources;
  for(unsigned int i = 0; i < batchsize; ++i)
    sources.push_back(s_distrib(prng));
  
  // Run the algorithm.
  auto algo_start = std::chrono::high_resolution_clock::now();
  run_bf(tr, batchsize, sources);
  auto t_algo = std::chrono::high_resolution_clock::now() - algo_start;
  
  std::cout << "time_mssp: "
	    << std::chrono::duration_cast<std::chrono::milliseconds>(t_algo).count() << std::endl;
}
